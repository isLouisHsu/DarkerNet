#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "prelu_layer.h"
#include "blas.h"
#include "hip/hip_runtime.h"
}

# if 1
__global__ void forward_prelu_layer_kernel(int n, int w, int h, int c, int g, float *input, float *weights, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;    // 所有通道索引

    int k = id % c;         // 某张量通道索引
    float alpha = weights[k / g];

    for(int i = 0; i < h*w; ++i){
        int idx = i + h*w * id;
        float val = input[idx];
        output[idx] = val < 0? alpha*val: val;
    }
}

extern "C" void  forward_prelu_layer_gpu(const layer l, network net)
{
    size_t n = l.c*l.batch;

    forward_prelu_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, l.w, l.h, l.c, l.groups, net.input_gpu, l.weights_gpu, l.output_gpu);
    check_error(hipPeekAtLastError());
}
#else
__global__ void prelu_kernel(float *input, float *output, float *alpha, int c, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x; // 像素索引
    int filter = blockIdx.y;    // 通道索引
    int batch = blockIdx.z;     // 数据索引
    int idx_n = filter / n;
    int index = (batch*c+filter)*size + offset;

    if(offset < size)
        output[index] = input[index] < 0 ? input[index]*alpha[idx_n] : input[index];
}

extern "C" void prelu_gpu(float *input, float *output, float *alpha, int batch, int c, int n, int size)
{
    dim3 dimGrid((size-1)/BLOCK + 1, c, batch);
    dim3 dimBlock(BLOCK, 1, 1);

    prelu_kernel<<<dimGrid, dimBlock>>>(input, output, alpha, c, n, size);
    check_error(hipPeekAtLastError());
}

extern "C" void forward_prelu_layer_gpu(layer l, network net){
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    prelu_gpu(net.input_gpu, l.output_gpu, l.weights_gpu, l.batch, l.c, l.groups, l.w*l.h);
}
#endif

void  backward_prelu_layer_gpu(layer l, network net)
{
    // TODO
}

void  update_prelu_layer_gpu(layer l, update_args a)
{
    // TODO
}

void pull_prelu_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
}

void push_prelu_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
}